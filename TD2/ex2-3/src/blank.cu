/***************************************************************************
															blank.cu
-------------------
begin                : Mon Oct 20 2010
email                : biri@univ-mlv.fr
***************************************************************************/

/***************************************************************************
*                                                                         *
*   This program is free software; you can redistribute it and/or modify  *
*   it under the terms of the GNU General Public License as published by  *
*   the Free Software Foundation; either version 2 of the License, or     *
*   (at your option) any later version.                                   *
*                                                                         *
***************************************************************************/

// includes, system
#include <cstdlib>
#include <iostream>
#include <cstdio>
#include <cstring>
#include <cmath>

// includes, project
#include "cudaSimpleUtils.hpp"
#include "imageTexture.hpp"

// includes, kernels
#include "blankKernel.cu"

const unsigned int NT = 20;
const unsigned int NB = 20;
const unsigned int N = NT * NB;

/// /////////////////////////////////////////////////////////////////////////////
//! Run a blank test for CUDA
//! renvoie le temps d'execution du kernel
/// /////////////////////////////////////////////////////////////////////////////
float runTest(float * h_src, float * h_dest_gpu) {
	// Variable initialization
	float * d_src;
	float * d_dest_gpu;
	const size_t ARRAY_BYTES = sizeof(float) * size_t(N);
	
	hipMalloc((void**)&d_src, ARRAY_BYTES);
	hipMalloc((void**)&d_dest_gpu, ARRAY_BYTES);
	
	// Get data from CPU
	hipMemcpy(d_src, h_src, ARRAY_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(d_dest_gpu, h_dest_gpu, ARRAY_BYTES, hipMemcpyHostToDevice);


	// setup execution parameters
	dim3  grid( 1, 1, 1);
	dim3  threads( 128, 1, 1);
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	// execute the kernel
	blankKernel<<< grid, threads>>>(d_src, d_dest_gpu);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float d_et;
	hipEventElapsedTime(&d_et, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	// Send data to CPU
	hipMemcpy(h_dest_gpu, d_dest_gpu, ARRAY_BYTES, hipMemcpyDeviceToHost);

	// Free memory
	hipFree(d_src);
	hipFree(d_dest_gpu);

	return d_et;
}

float runLuminance(unsigned char* const h_rgbaImage, unsigned char* const h_greyImage, int numRows, int numCols) {

	unsigned char* d_src;
	unsigned char* d_res;

	const unsigned int threadDimX = 16;
	const unsigned int threadDimY = 16;
	const unsigned int blockDimX = numRows/threadDimX;
	const unsigned int blockDimY = numCols/threadDimY;

	const size_t ARRAY_BYTES = sizeof(unsigned char) * size_t(3*numRows*numCols);

	hipMalloc((void**)&d_src, ARRAY_BYTES);
	hipMalloc((void**)&d_res, ARRAY_BYTES);

	hipMemcpy(d_src, h_rgbaImage, ARRAY_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(d_res, h_greyImage, ARRAY_BYTES, hipMemcpyHostToDevice);

	dim3  grid( blockDimX, blockDimY, 1);
	dim3  threads( threadDimX, threadDimY, 1);
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	luminanceKernel<<<grid, threads>>>(d_src, d_res, numRows, numCols);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float d_et;
	hipEventElapsedTime(&d_et, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipMemcpy(h_greyImage, d_res, ARRAY_BYTES, hipMemcpyDeviceToHost);

	hipFree(d_src);
	hipFree(d_res);

	return d_et;
}

float* runAdditionOf2Matrix(float* h_mat1, float* h_mat2) {

	float* d_mat1;
	float* d_mat2;
	float* d_matRes;

	const unsigned int threadDimX = 3;
	const unsigned int threadDimY = 3;

	const size_t ARRAY_BYTES = sizeof(float) * 9;

	hipMalloc((void**)&d_mat1, ARRAY_BYTES);
	hipMalloc((void**)&d_mat2, ARRAY_BYTES);
	hipMalloc((void**)&d_matRes, ARRAY_BYTES);

	hipMemcpy(d_mat1, h_mat1, ARRAY_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(d_mat2, h_mat2, ARRAY_BYTES, hipMemcpyHostToDevice);

	dim3  grid(1, 1, 1);
	dim3  threads(threadDimX, threadDimY, 1);
	additionOf2MatrixKernel<<<grid, threads>>>(d_mat1, d_mat2, d_matRes);

	float* h_matRes = new float[9];
	hipMemcpy(h_matRes, d_matRes, ARRAY_BYTES, hipMemcpyDeviceToHost);

	hipFree(d_mat1);
	hipFree(d_mat2);
	hipFree(d_matRes);

	return h_matRes;
}
