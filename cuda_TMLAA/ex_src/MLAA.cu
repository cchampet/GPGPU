/***************************************************************************
                              vectorTest.cu
                           -------------------
begin                : Mon Oct 20 2010
email                : biri@univ-mlv.fr
***************************************************************************/

/***************************************************************************
*                                                                         *
*   This program is free software; you can redistribute it and/or modify  *
*   it under the terms of the GNU General Public License as published by  *
*   the Free Software Foundation; either version 2 of the License, or     *
*   (at your option) any later version.                                   *
*                                                                         *
***************************************************************************/

// includes, system
#include <cstdlib>
#include <iostream>
#include <cstdio>
#include <cstring>
#include <cmath>

// includes, project
#include <cudaSimpleUtils.hpp>

// includes, kernels
#include "MLAAKernel.cu"
#include "main.hpp"


/// /////////////////////////////////////////////////////////////////////////////
//! Run CUDA
/// /////////////////////////////////////////////////////////////////////////////
float cudaRunDiscontinuity(unsigned char *idata, unsigned char* odata, unsigned int sz_thread_per_block, unsigned int num_blocksx, unsigned int num_blocksy) {
	// compute size
	unsigned int width = num_blocksx * sz_thread_per_block ;
	unsigned int height = num_blocksy * sz_thread_per_block;

	unsigned int mem_size = 3*width*height*sizeof(unsigned char);
	unsigned int mem_size_buffer = 3*width*height*sizeof(float);
	// Génération des timers
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// allocate device memory
	unsigned char* d_idata;
	if (!(hipMalloc( (void**) &d_idata, mem_size) == hipSuccess)) {
		std::cerr<<"Erreur allocating memory in device ! Aborting!"<<std::endl;
		exit(1);
	}
	// copy host memory to device
	if (!(hipMemcpy( d_idata, idata, mem_size, hipMemcpyHostToDevice) == hipSuccess)) {
		std::cerr<<"Erreur copying memory in device ! Aborting!"<<std::endl;
		exit(1);
	}

	// allocate device memory for result
	float* d_odataXYZ;
	if (!(hipMalloc( (void**) &d_odataXYZ, mem_size_buffer) == hipSuccess)) {
		std::cerr<<"Erreur allocating memory in device ! Aborting!"<<std::endl;
		exit(1);
	}
	float* d_odataLab;
	if (!(hipMalloc( (void**) &d_odataLab, mem_size_buffer) == hipSuccess)) {
		std::cerr<<"Erreur allocating memory in device ! Aborting!"<<std::endl;
		exit(1);
	}
	unsigned char* d_odata;
	if (!(hipMalloc( (void**) &d_odata, mem_size) == hipSuccess)) {
		std::cerr<<"Erreur allocating memory in device ! Aborting!"<<std::endl;
		exit(1);
	}

	// setup execution parameters
	dim3  grid( num_blocksx, num_blocksy, 1);
	dim3  threads( sz_thread_per_block, sz_thread_per_block, 1);

	// execute the kernels
	hipEventRecord(start, 0);
	DetectDiscontinuity<<< grid, threads >>>(d_idata, d_odataXYZ, d_odataLab, d_odata);
	hipEventRecord(stop, 0);

	// check if kernel execution generated and error
	cutilCheckMsg("Kernel execution failed","MLAA.cu",135);
	hipEventSynchronize(stop);

	// copy result from device to host
	hipMemcpy(odata, d_odata, mem_size, hipMemcpyDeviceToHost);

	// GESTION DU TEMPS
	float elapsed_time;
	hipEventElapsedTime(&elapsed_time, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipFree(d_idata);
	hipFree(d_odataXYZ);
	hipFree(d_odataLab);
	hipFree(d_odata);

	hipDeviceReset();

	return elapsed_time;
}

float cudaRunHorizontalDoubling(unsigned char *idata, unsigned char* odata_l, unsigned char* odata_r, unsigned int sz_thread_per_blockx, unsigned int sz_thread_per_blocky, unsigned int num_blocksx, unsigned int num_blocksy) {
	// compute size
	unsigned int width = num_blocksx * sz_thread_per_blockx;
	unsigned int height = num_blocksy * sz_thread_per_blocky;

	unsigned int mem_size = 3*width*height*sizeof(unsigned char);
	// Génération des timers
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// allocate device memory
	unsigned char* d_idata;
	if (!(hipMalloc( (void**) &d_idata, mem_size) == hipSuccess)) {
		std::cerr<<"Erreur allocating memory in device ! Aborting!"<<std::endl;
		exit(1);
	}
	// copy host memory to device
	if (!(hipMemcpy( d_idata, idata, mem_size, hipMemcpyHostToDevice) == hipSuccess)) {
		std::cerr<<"Erreur copying memory in device ! Aborting!"<<std::endl;
		exit(1);
	}

	// allocate device memory for result
	unsigned char* d_odata_l;
	if (!(hipMalloc( (void**) &d_odata_l, mem_size) == hipSuccess)) {
		std::cerr<<"Erreur allocating memory in device ! Aborting!"<<std::endl;
		exit(1);
	}

	unsigned char* d_odata_r;
	if (!(hipMalloc( (void**) &d_odata_r, mem_size) == hipSuccess)) {
		std::cerr<<"Erreur allocating memory in device ! Aborting!"<<std::endl;
		exit(1);
	}

	// setup execution parameters
	dim3  grid( num_blocksx, num_blocksy, 1);
	dim3  threads( sz_thread_per_blockx, sz_thread_per_blocky, 1);

	// execute the kernels
	hipEventRecord(start, 0);
	RecursiveDoublingHorizontal<<< grid, threads >>>(d_idata, d_odata_l, d_odata_r);
	hipEventRecord(stop, 0);

	// check if kernel execution generated and error
	cutilCheckMsg("Kernel execution failed","MLAA.cu",135);
	hipEventSynchronize(stop);

	// copy result from device to host
	hipMemcpy(odata_l, d_odata_l, mem_size, hipMemcpyDeviceToHost);
	hipMemcpy(odata_r, d_odata_r, mem_size, hipMemcpyDeviceToHost);

	// GESTION DU TEMPS
	float elapsed_time;
	hipEventElapsedTime(&elapsed_time, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipFree(d_idata);
	hipFree(d_odata_l);
	hipFree(d_odata_r);

	hipDeviceReset();

	return elapsed_time;
}

float cudaRunVerticalDoubling(unsigned char *idata, unsigned char* odata_t, unsigned char* odata_b, unsigned int sz_thread_per_blockx, unsigned int sz_thread_per_blocky, unsigned int num_blocksx, unsigned int num_blocksy) {
	// compute size
	unsigned int width = num_blocksx * sz_thread_per_blockx;
	unsigned int height = num_blocksy * sz_thread_per_blocky;

	unsigned int mem_size = 3*width*height*sizeof(unsigned char);
	// Génération des timers
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// allocate device memory
	unsigned char* d_idata;
	if (!(hipMalloc( (void**) &d_idata, mem_size) == hipSuccess)) {
		std::cerr<<"Erreur allocating memory in device ! Aborting!"<<std::endl;
		exit(1);
	}
	// copy host memory to device
	if (!(hipMemcpy( d_idata, idata, mem_size, hipMemcpyHostToDevice) == hipSuccess)) {
		std::cerr<<"Erreur copying memory in device ! Aborting!"<<std::endl;
		exit(1);
	}

	// allocate device memory for result
	unsigned char* d_odata_t;
	if (!(hipMalloc( (void**) &d_odata_t, mem_size) == hipSuccess)) {
		std::cerr<<"Erreur allocating memory in device ! Aborting!"<<std::endl;
		exit(1);
	}

	unsigned char* d_odata_b;
	if (!(hipMalloc( (void**) &d_odata_b, mem_size) == hipSuccess)) {
		std::cerr<<"Erreur allocating memory in device ! Aborting!"<<std::endl;
		exit(1);
	}

	// setup execution parameters
	dim3  grid( num_blocksx, num_blocksy, 1);
	dim3  threads( sz_thread_per_blockx, sz_thread_per_blocky, 1);

	// execute the kernels
	hipEventRecord(start, 0);
	RecursiveDoublingVertical<<< grid, threads >>>(d_idata, d_odata_t, d_odata_b);
	hipEventRecord(stop, 0);

	// check if kernel execution generated and error
	cutilCheckMsg("Kernel execution failed","MLAA.cu",135);
	hipEventSynchronize(stop);

	// copy result from device to host
	hipMemcpy(odata_t, d_odata_t, mem_size, hipMemcpyDeviceToHost);
	hipMemcpy(odata_b, d_odata_b, mem_size, hipMemcpyDeviceToHost);

	// GESTION DU TEMPS
	float elapsed_time;
	hipEventElapsedTime(&elapsed_time, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipFree(d_idata);
	hipFree(d_odata_t);
	hipFree(d_odata_b);

	hipDeviceReset();

	return elapsed_time;
}

float cudaRunBlending(unsigned char *idata, unsigned char *idata_hl, unsigned char *idata_vt, unsigned char* odata, unsigned int sz_thread_per_blockx, unsigned int sz_thread_per_blocky, unsigned int num_blocksx, unsigned int num_blocksy) {
	// compute size
	unsigned int width = num_blocksx * sz_thread_per_blockx;
	unsigned int height = num_blocksy * sz_thread_per_blocky;

	unsigned int mem_size = 3*width*height*sizeof(unsigned char);
	unsigned int mem_size_weight = width*height*sizeof(float);
	// Génération des timers
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// allocate device memory
	unsigned char* d_idata;
	if (!(hipMalloc( (void**) &d_idata, mem_size) == hipSuccess)) {
		std::cerr<<"Erreur allocating memory in device ! Aborting!"<<std::endl;
		exit(1);
	}
	// copy host memory to device
	if (!(hipMemcpy( d_idata, idata, mem_size, hipMemcpyHostToDevice) == hipSuccess)) {
		std::cerr<<"Erreur copying memory in device ! Aborting!"<<std::endl;
		exit(1);
	}

	// allocate device memory
	unsigned char* d_idata_hl;
	if (!(hipMalloc( (void**) &d_idata_hl, mem_size) == hipSuccess)) {
		std::cerr<<"Erreur allocating memory in device ! Aborting!"<<std::endl;
		exit(1);
	}
	// copy host memory to device
	if (!(hipMemcpy( d_idata_hl, idata_hl, mem_size, hipMemcpyHostToDevice) == hipSuccess)) {
		std::cerr<<"Erreur copying memory in device ! Aborting!"<<std::endl;
		exit(1);
	}

	unsigned char* d_idata_vt;
	if (!(hipMalloc( (void**) &d_idata_vt, mem_size) == hipSuccess)) {
		std::cerr<<"Erreur allocating memory in device ! Aborting!"<<std::endl;
		exit(1);
	}
	// copy host memory to device
	if (!(hipMemcpy( d_idata_vt, idata_vt, mem_size, hipMemcpyHostToDevice) == hipSuccess)) {
		std::cerr<<"Erreur copying memory in device ! Aborting!"<<std::endl;
		exit(1);
	}

	// allocate device memory for result
	float* d_odata_wl;
	if (!(hipMalloc( (void**) &d_odata_wl, mem_size_weight) == hipSuccess)) {
		std::cerr<<"Erreur allocating memory in device ! Aborting!"<<std::endl;
		exit(1);
	}
	float* d_odata_wt;
	if (!(hipMalloc( (void**) &d_odata_wt, mem_size_weight) == hipSuccess)) {
		std::cerr<<"Erreur allocating memory in device ! Aborting!"<<std::endl;
		exit(1);
	}

	unsigned char* d_odata_blending_horizontal;
	if (!(hipMalloc( (void**) &d_odata_blending_horizontal, mem_size) == hipSuccess)) {
		std::cerr<<"Erreur allocating memory in device ! Aborting!"<<std::endl;
		exit(1);
	}

	unsigned char* d_odata;
	if (!(hipMalloc( (void**) &d_odata, mem_size) == hipSuccess)) {
		std::cerr<<"Erreur allocating memory in device ! Aborting!"<<std::endl;
		exit(1);
	}

	// setup execution parameters
	dim3  grid( num_blocksx, num_blocksy, 1);
	dim3  threads( sz_thread_per_blockx, sz_thread_per_blocky, 1);

	// execute the kernels
	hipEventRecord(start, 0);
	ComputeWeightHorizontal<<< grid, threads >>>(d_idata_hl, d_odata_wl);
	BlendingHorizontal<<< grid, threads >>>(d_idata, d_idata_hl, d_odata_wl, d_odata_blending_horizontal);
	
	ComputeWeightVertical<<< grid, threads >>>(d_idata_vt, d_odata_wt);
	BlendingVertical<<< grid, threads >>>(d_odata_blending_horizontal, d_idata_vt, d_odata_wt, d_odata);

	hipEventRecord(stop, 0);

	// check if kernel execution generated and error
	cutilCheckMsg("Kernel execution failed","MLAA.cu",135);
	hipEventSynchronize(stop);

	// copy result from device to host
	hipMemcpy(odata, d_odata, mem_size, hipMemcpyDeviceToHost);

	// GESTION DU TEMPS
	float elapsed_time;
	hipEventElapsedTime(&elapsed_time, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipFree(d_idata);
	hipFree(d_idata_hl);
	hipFree(d_idata_vt);
	hipFree(d_odata_wl);
	hipFree(d_odata_wt);
	hipFree(d_odata_blending_horizontal);
	hipFree(d_odata);

	hipDeviceReset();

	return elapsed_time;
}