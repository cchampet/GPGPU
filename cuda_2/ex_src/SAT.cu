/***************************************************************************
                              vectorTest.cu
                           -------------------
begin                : Mon Oct 20 2010
email                : biri@univ-mlv.fr
***************************************************************************/

/***************************************************************************
*                                                                         *
*   This program is free software; you can redistribute it and/or modify  *
*   it under the terms of the GNU General Public License as published by  *
*   the Free Software Foundation; either version 2 of the License, or     *
*   (at your option) any later version.                                   *
*                                                                         *
***************************************************************************/

// includes, system
#include <cstdlib>
#include <iostream>
#include <cstdio>
#include <cstring>
#include <cmath>

// includes, project
#include <cudaSimpleUtils.hpp>

// includes, kernels
#include "SATKernel.cu"
#include "main.hpp"


/// /////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
/// /////////////////////////////////////////////////////////////////////////////
float runTest(unsigned int *idata, unsigned int* odata, unsigned int* oHisto, unsigned int sz_thread_per_block, unsigned int num_blocksx, unsigned int num_blocksy) {
	// TO DO : compute size...
	unsigned int width = num_blocksx * sz_thread_per_block ;
	unsigned int height = num_blocksy * sz_thread_per_block;

	unsigned int mem_size = 3*width*height*sizeof(unsigned int);
	// Génération des timers ...
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// allocate device memory
	unsigned int* d_idata;
	if (!(hipMalloc( (void**) &d_idata, mem_size) == hipSuccess)) {
		std::cerr<<"Erreur allocating memory in device ! Aborting!"<<std::endl;
		exit(1);
	}
	// copy host memory to device
	if (!(hipMemcpy( d_idata, idata, mem_size, hipMemcpyHostToDevice) == hipSuccess)) {
		std::cerr<<"Erreur copying memory in device ! Aborting!"<<std::endl;
		exit(1);
	}

	// allocate device memory for result
	unsigned int* d_odata;
	if (!(hipMalloc( (void**) &d_odata, mem_size) == hipSuccess)) {
		std::cerr<<"Erreur allocating memory in device ! Aborting!"<<std::endl;
		exit(1);
	}

	// allocate device memory for historgramme result
	unsigned int* d_oHisto;
	if (!(hipMalloc( (void**) &d_oHisto, 256*sizeof(unsigned int)) == hipSuccess)) {
		std::cerr<<"Erreur allocating memory in device ! Aborting!"<<std::endl;
		exit(1);
	}
	// copy host memory to device
	if (!(hipMemcpy( d_oHisto, oHisto, 256*sizeof(unsigned int), hipMemcpyHostToDevice) == hipSuccess)) {
		std::cerr<<"Erreur copying memory in device ! Aborting!"<<std::endl;
		exit(1);
	}

	// setup execution parameters
	dim3  grid( num_blocksx, num_blocksy, 1);
	dim3  threads( sz_thread_per_block, sz_thread_per_block, 1);

	// execute the kernels
	hipEventRecord(start, 0);
	LumKernel<<< grid, threads >>>( d_idata, d_odata);
	HistoKernel<<< grid, threads >>>( d_odata, d_oHisto);
	hipEventRecord(stop, 0);

	// check if kernel execution generated and error
	cutilCheckMsg("Kernel execution failed","SAT.cu",135);
	hipEventSynchronize(stop);

	// copy result from device to host
	hipMemcpy( odata, d_odata, mem_size, hipMemcpyDeviceToHost);
	hipMemcpy( oHisto, d_oHisto, 256*sizeof(unsigned int), hipMemcpyDeviceToHost);

	// GESTION DU TEMPS
	float elapsed_time;
	hipEventElapsedTime(&elapsed_time, start, stop);
	hipEventDestroy(start);hipEventDestroy(stop);

	hipFree(d_idata);
	hipFree(d_odata);
	hipFree(d_oHisto);

	hipDeviceReset();

	return elapsed_time;
}

float runGauss(unsigned int *idata, unsigned int* odata, unsigned int sz_thread_per_block, unsigned int num_blocksx, unsigned int num_blocksy) {
	// TO DO : compute size...
	unsigned int width = num_blocksx * sz_thread_per_block ;
	unsigned int height = num_blocksy * sz_thread_per_block;

	unsigned int mem_size = width*height*sizeof(unsigned int);
	// Génération des timers ...
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// allocate device memory
	unsigned int* d_idata;
	if (!(hipMalloc( (void**) &d_idata, mem_size) == hipSuccess)) {
		std::cerr<<"Erreur allocating memory in device ! Aborting!"<<std::endl;
		exit(1);
	}
	// copy host memory to device
	if (!(hipMemcpy( d_idata, idata, mem_size, hipMemcpyHostToDevice) == hipSuccess)) {
		std::cerr<<"Erreur copying memory in device ! Aborting!"<<std::endl;
		exit(1);
	}

	// allocate device memory for result
	unsigned int* d_odata;
	if (!(hipMalloc( (void**) &d_odata, mem_size) == hipSuccess)) {
		std::cerr<<"Erreur allocating memory in device ! Aborting!"<<std::endl;
		exit(1);
	}

	// setup execution parameters
	dim3  grid( num_blocksx, num_blocksy, 1);
	dim3  threads( sz_thread_per_block, sz_thread_per_block, 1);

	// execute the kernel
	hipEventRecord(start, 0);
	GaussKernel<<< grid, threads >>>( d_idata, d_odata);
	hipEventRecord(stop, 0);

	// check if kernel execution generated and error
	cutilCheckMsg("Kernel execution failed","SAT.cu",135);
	hipEventSynchronize(stop);

	// copy result from device to host
	hipMemcpy( odata, d_odata, mem_size, hipMemcpyDeviceToHost);

	// GESTION DU TEMPS
	float elapsed_time;
	hipEventElapsedTime(&elapsed_time, start, stop);
	hipEventDestroy(start);hipEventDestroy(stop);

	hipFree(d_idata);
	hipFree(d_odata);

	hipDeviceReset();

	return elapsed_time;
}
